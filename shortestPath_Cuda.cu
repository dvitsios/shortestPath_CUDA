#include "hip/hip_runtime.h"
/*
 *  Copyright (C) 2009 by Vitsios Dimitrios
 *
 *  Permission is hereby granted, free of charge, to any person obtaining a copy
 *  of this software and associated documentation files (the "Software"), to deal
 *  in the Software without restriction, including without limitation the rights
 *  to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 *  copies of the Software, and to permit persons to whom the Software is
 *  furnished to do so, subject to the following conditions:
 *
 *  The above copyright notice and this permission notice shall be included in
 *  all copies or substantial portions of the Software.
 *
 *  THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 *  IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 *  FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 *  AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 *  LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 *  OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
 *  THE SOFTWARE.
 */

/**
 * Shortest path,
 * parallel implementation
 * using CUDA
 */


#include <stdio.h>
#include <sys/types.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <math.h>
#include <float.h>
#define N 512


__global__ void shortest_path( float *c, float *C , int k, int iter, int *path, int counter, int L, int K)
{

__shared__ float C_th[N]; 
__shared__ int MIN_IDX[N];

  unsigned int s;

  int index = blockIdx.x * N + threadIdx.x;

  MIN_IDX[threadIdx.x] = threadIdx.x;

  int offset = (k*k+iter)*N*N;

  C_th[threadIdx.x] = C[threadIdx.x] + c[index + offset];
  
  __syncthreads();

  
  for(s=blockDim.x/2;s>32;s>>=1){
	if(threadIdx.x < s){
		if( C_th[MIN_IDX[threadIdx.x]] > C_th[MIN_IDX[threadIdx.x + s]]){
			MIN_IDX[threadIdx.x] = MIN_IDX[threadIdx.x + s];
		}
	}
	__syncthreads();
  }	

  if(threadIdx.x < 32){
	if(N > 32){	
		if( C_th[MIN_IDX[threadIdx.x]] > C_th[MIN_IDX[threadIdx.x + 32]]){
				MIN_IDX[threadIdx.x] = MIN_IDX[threadIdx.x + 32];
			}
	}
	if(N > 16){	
		if( C_th[MIN_IDX[threadIdx.x]] > C_th[MIN_IDX[threadIdx.x + 16]]){
				MIN_IDX[threadIdx.x] = MIN_IDX[threadIdx.x + 16];
			}
 	}
	if(N > 8){	
		if( C_th[MIN_IDX[threadIdx.x]] > C_th[MIN_IDX[threadIdx.x + 8]]){
				MIN_IDX[threadIdx.x] = MIN_IDX[threadIdx.x + 8];
			}
	}
	if(N > 4){	
		if( C_th[MIN_IDX[threadIdx.x]] > C_th[MIN_IDX[threadIdx.x + 4]]){
				MIN_IDX[threadIdx.x] = MIN_IDX[threadIdx.x + 4];
			}
 	}
	if(N > 2){	
		if( C_th[MIN_IDX[threadIdx.x]] > C_th[MIN_IDX[threadIdx.x + 2]]){
				MIN_IDX[threadIdx.x] = MIN_IDX[threadIdx.x + 2];
			}
	}
		if( C_th[MIN_IDX[threadIdx.x]] > C_th[MIN_IDX[threadIdx.x + 1]]){
				MIN_IDX[threadIdx.x] = MIN_IDX[threadIdx.x + 1];
			}
  }

  __syncthreads();

  if(threadIdx.x == 0){
  	C[blockIdx.x] = C_th[MIN_IDX[0]]; 
	path[blockIdx.x*L + counter] = MIN_IDX[0];	
  }

}


int main(int argc, char* argv[])
{

FILE *f_path;
clock_t start, end;
int i,j,r=0, k=1, iter, counter=0, *path_host, *path, L, K;
float *c_host, *c, *C_host, *C_host_L, *C;  

printf("Type the number of levels of the graph (L):       [ < 2048]\n\n");
scanf("%d",&L);

printf("Type the value of 'K':       \n\n");
scanf("%d",&K);

hipStream_t stream[2];
hipStreamCreate(&stream[0]);
hipStreamCreate(&stream[1]);


C_host = (float *)malloc( N * sizeof(float) );
C_host_L = (float *)malloc( N * sizeof(float) );
path_host = (int *)malloc( L * N * sizeof(int) );
path = (int *)malloc( L * N * sizeof(int) );


int size_c = L * N * N * sizeof(float);
hipHostMalloc((void**)&c_host, size_c);

srand(5);

	//initialiZe c_host[][] matrix...
	for(i=0; i<L*N*N; i++)
             c_host[i] = (float)(rand() % 1000 + 10)/100; //supposing that edges have costs from  0.1 to 109

	//initialiZe C_host[][] matrix...		 
	for(i=0; i<N; i++)
             C_host[i] = (float)(rand() % 1000 + 10)/100;
	//initialiZe random C_host matrix for the last but one level
	for(i=0; i<N; i++)
             C_host_L[i] = (float)(rand() % 1000 + 10)/100;

start=clock();

int size1 = N * N * sizeof(float);
hipMalloc((void**)&c,2*K*size1);
hipMemcpy( c, c_host, K*size1, hipMemcpyHostToDevice );

int size2 = N * sizeof(float);
hipMalloc((void**)&C,size2);
hipMemcpy( C, C_host, size2, hipMemcpyHostToDevice );

int size3 = N * L* sizeof(int);
hipMalloc((void**)&path,size3);
hipMemcpy( path, path_host, size3, hipMemcpyHostToDevice );
    
printf("GPU computing started!\n");

for(r=1; r<(L/K); r++){
		
		
	hipMemcpyAsync( c+k*K*N*N, c_host+r*K*N*N, K*size1, hipMemcpyHostToDevice, stream[0] );

	for(iter=0;iter<K;iter++){
		shortest_path<<< N, N, 0, stream[1] >>>( c, C , !k, iter, path, counter, L, K);
		counter++;
	} 

	hipDeviceSynchronize();	 		
	k == 0 ? k = 1 : k = 0;

}

for(iter=0;iter<K;iter++){
	
	shortest_path<<< N, N, 0, stream[0] >>>( c, C , !k, iter, path, counter, L, K);
	counter++;
}

hipMemcpy(C_host, C, size2, hipMemcpyDeviceToHost);
hipMemcpy(path_host, path, size3, hipMemcpyDeviceToHost);


end=clock();




float total_min = C_host[0]+C_host_L[0];
int total_min_idx = 0;
for(i=1; i<N; i++){
	if( C_host[i] + C_host_L[i]< total_min){
		total_min = C_host[i]+ C_host_L[i];
		total_min_idx = i;
	}
}

printf("\nTotal min = %f", total_min);
printf("\nTotal min INDEX = %d", total_min_idx);

f_path = fopen("path.txt","w");
             
printf("\n\n*** path_host ***\n");
	for(j=0; j<L; j++){
             printf("%d ", path_host[ total_min_idx*L + j ]);             
	     	 fprintf(f_path,"%d", path_host[ total_min_idx*L + j ]);
	}


    printf("\n\n*******************************************************************************");
    printf("\nTotal time elapsed for transfering the data and computing in GPU: %d ms",(end-start)*1000/CLOCKS_PER_SEC);	


scanf("%d",&i);
return EXIT_SUCCESS;
}
